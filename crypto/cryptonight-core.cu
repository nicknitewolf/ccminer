#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>

#include "cryptonight.h"
#define LONG_SHL_IDX 19U
#define LONG_LOOPS32 0x80000U

#include "cn_aes.cuh"

__global__
//__launch_bounds__(128, 9) // 56 registers
void cryptonight_core_gpu_phase1(const uint32_t threads, uint32_t * long_state, uint32_t * const ctx_state, uint32_t * ctx_key1)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];

	cn_aes_gpu_init(sharedMemory);

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3;
	const uint32_t sub = (threadIdx.x & 7) << 2; // 0 4 8 ... 28

	if(thread < threads)
	{
		const uint32_t long_oft = (thread << LONG_SHL_IDX) + sub;
		ulonglong2 text = AS_UL2(&ctx_state[thread * 52U + sub + 16U]);

		const uint32_t* ctx_key = &ctx_key1[thread * 40U];
		uint32_t key[40];
		#pragma unroll 10 // copy 160 bytes
		for (uint32_t i = 0; i < 40U; i += 4U)
			AS_UINT4(&key[i]) = AS_UINT4(&ctx_key[i]);

		__threadfence_block();

		for(uint32_t i = 0; i < LONG_LOOPS32; i += 32U) {
			cn_aes_pseudo_round_mut(sharedMemory, (uint32_t*) &text, key);
			AS_UL2(&long_state[long_oft + i]) = text;
		}
	}
}

static __forceinline__ __device__ ulonglong2 operator ^ (const ulonglong2 &a, const ulonglong2 &b) {
	return make_ulonglong2(a.x ^ b.x, a.y ^ b.y);
}
static __forceinline__ __device__ uint4 operator ^ (const uint4 &a, const uint4 &b) {
	return make_uint4(a.x ^ b.x, a.y ^ b.y, a.z ^ b.z, a.w ^ b.w);
}

__device__ __forceinline__ ulonglong2 cuda_mul128(const uint64_t multiplier, const uint64_t multiplicand)
{
	ulonglong2 product;
	product.x = __umul64hi(multiplier, multiplicand);
	product.y = multiplier * multiplicand;
	return product;
}

static __forceinline__ __device__ void operator += (ulonglong2 &a, const ulonglong2 b) {
	a.x += b.x; a.y += b.y;
}

#undef MUL_SUM_XOR_DST
__device__ __forceinline__ void MUL_SUM_XOR_DST(const uint64_t m, uint4 &a, void* far_dst)
{
	ulonglong2 d = AS_UL2(far_dst);
	ulonglong2 p = cuda_mul128(m, d.x);
	p += AS_UL2(&a);
	AS_UL2(&a) = p ^ d;
	AS_UL2(far_dst) = p;
}

__global__
#if __CUDA_ARCH__ >= 500
//__launch_bounds__(128,12) /* force 40 regs to allow -l ...x32 */
#endif
void cryptonight_core_gpu_phase2(const uint32_t threads, const uint32_t bfactor, const uint32_t partidx,
	uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];

//	cn_aes_gpu_init(sharedMemory);
//	__syncthreads();

	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	if (thread < threads)
	{
		const uint32_t batchsize = ITER >> (2U + bfactor);
		const uint32_t start = partidx * batchsize;
		const uint32_t end = start + batchsize;
		const uint32_t longptr = thread << LONG_SHL_IDX;

		uint32_t * long_state = &d_long_state[longptr];

		void * ctx_a = (void*)(&d_ctx_a[thread << 2U]);
		void * ctx_b = (void*)(&d_ctx_b[thread << 2U]);
		uint4 A = AS_UINT4(ctx_a); // ld.global.u32.v4
		uint4 B = AS_UINT4(ctx_b);

		for (int i = start; i < end; i++) // end = 262144
		{
			uint4 C;

			uint32_t j = (A.x >> 2) & E2I_MASK2;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &C);
			AS_UINT4(&long_state[j]) = C ^ B; // // st.global.u32.v4
			MUL_SUM_XOR_DST((AS_UL2(&C)).x, A, &long_state[(C.x >> 2U) & E2I_MASK2]);

			j = (A.x >> 2) & E2I_MASK2;
			cn_aes_single_round_b((uint8_t*)sharedMemory, &long_state[j], A, &B);
			AS_UINT4(&long_state[j]) = C ^ B;
			MUL_SUM_XOR_DST((AS_UL2(&B)).x, A, &long_state[(B.x >> 2U) & E2I_MASK2]);
		}

		if (bfactor) {
			AS_UINT4(ctx_a) = A;
			AS_UINT4(ctx_b) = B;
		}
	}
}

__global__
void cryptonight_core_gpu_phase3(const uint32_t threads, const uint32_t * __restrict__ long_state, uint32_t * ctx_state, uint32_t * __restrict__ ctx_key2)
{
	__shared__ __align__(16) uint32_t sharedMemory[1024];

	//cn_aes_gpu_init(sharedMemory);

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 3U;
	const uint32_t sub = (threadIdx.x & 7U) << 2U;

	if(thread < threads)
	{
		const uint32_t long_oft = (thread << LONG_SHL_IDX) + sub;
		const uint32_t st_oft = thread * 52U + sub + 16U;

		ulonglong2 text = AS_UL2(&ctx_state[st_oft]);

		// copy 160 bytes
		uint32_t key[40];
		const uint32_t* ctx_key = &ctx_key2[thread * 40U];
		#pragma unroll 10
		for (uint32_t i = 0; i < 40U; i += 4U)
			AS_UL2(&key[i]) = AS_UL2(&ctx_key[i]);

		//__syncthreads();
		for(uint32_t i = 0; i < LONG_LOOPS32; i += 32U)
		{
			ulonglong2 st = AS_UL2(&long_state[long_oft + i]);
			text = text ^ st;
			cn_aes_pseudo_round_mut(sharedMemory, (uint32_t*) (&text), key);
		}

		AS_UL2(&ctx_state[st_oft]) = text;
	}
}

extern int device_bfactor[MAX_GPUS];

__host__
void cryptonight_core_cpu_hash(int thr_id, int blocks, int threads, uint32_t *d_long_state, uint64_t *d_ctx_state,
	uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2)
{
	dim3 grid(blocks);
	dim3 block(threads);
	dim3 block2(threads << 1);
	dim3 block4(threads << 2);
	dim3 block8(threads << 3);

	const uint32_t bfactor = (uint32_t) device_bfactor[thr_id];
	const uint32_t partcount = 1 << bfactor;
	const uint32_t throughput = (uint32_t) (blocks*threads);

	const int bsleep = bfactor ? 100 : 0;
	const int dev_id = device_map[thr_id];
	int i;

	cryptonight_core_gpu_phase1 <<<grid, block8, 4096>>> (throughput, d_long_state, (uint32_t*)d_ctx_state, d_ctx_key1);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
	if(partcount > 1) usleep(bsleep);

	for(i = 0; i < partcount; i++)
	{
		dim3 b = device_sm[dev_id] >= 300 ? block4 : block;
		cryptonight_core_gpu_phase2 <<<grid, b, 4096>>> (throughput, bfactor, i, d_long_state, d_ctx_a, d_ctx_b);
		exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
		if(partcount > 1) usleep(bsleep);
	}

	cryptonight_core_gpu_phase3 <<<grid, block8, 4096>>> (throughput, d_long_state, (uint32_t*)d_ctx_state, d_ctx_key2);
	exit_if_cudaerror(thr_id, __FUNCTION__, __LINE__);
}
